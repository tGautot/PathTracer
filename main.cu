#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include "headers/vec3.h"
#include "headers/ray.h"
#include "headers/hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>


__device__ vec3 color(const ray& r) {
   vec3 unit_direction = r.direction().normalized();
   float t = 0.5f*(unit_direction.y() + 1.0f);
   return ((1.0f-t)*vec3(1.0, 1.0, 1.0) + t*vec3(0.5, 0.7, 1.0)) * 255.0f;
}

__global__ 
void render(int szx, int szy, vec3 *colors, vec3 lower_left, vec3 du, vec3 dv){
    int stride_x = blockDim.x, stride_y = blockDim.y;

    int pixid;
    for(int i = threadIdx.x; i < szx; i+= stride_x ){
        for(int j = threadIdx.y; j < szy; j+= stride_y ){
            ray r = ray(vec3(0,0,0), lower_left+i*du+j*dv);
            pixid = (j*szx+i);
            colors[pixid] = color(r);
        }
    }

}

int main(){
    int nx = 600, ny = 600;
    int num_pix = nx*ny;

    size_t color_arr_size = num_pix*sizeof(vec3);
    vec3 *colors;
    checkCudaErrors(hipMallocManaged((void**)&colors, color_arr_size));
    
    dim3 threadDim(8,8);
    render<<<1,threadDim>>>(nx,ny,colors, vec3(-300, -300, -10), vec3(1,0,0), vec3(0,1,0));
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    std::cout << "P3\n" << nx << " " << ny << "\n255\n";
    for (int j = ny-1; j >= 0; j--) {
        for (int i = 0; i < nx; i++) {
            size_t pixid = j*nx + i;
            
            std::cout << (int) colors[pixid].x() << " " << (int) colors[pixid+1].y() << " " << (int) colors[pixid+2].z() << "\n";
        }
    }
    checkCudaErrors(hipFree(colors));
}